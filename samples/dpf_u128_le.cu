#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>
#include <dpf.h>

#define kThreadsPerBlock 256
#define kIterNum 1000000

#define kSeed 114514
#define kAlpha 107
#define kAlphaBitlen 16
#define kBeta 604

extern void prg_init(const uint8_t *state, int state_len);

static inline double get_time() {
  struct timespec ts;
  clock_gettime(CLOCK_REALTIME, &ts);
  return ts.tv_sec + ts.tv_nsec * 1.0e-9;
}

__global__ void gen_kernel(
  uint8_t *cw_np1_dev, uint8_t *cws_dev, const uint8_t *alpha_dev, const uint8_t *beta_dev, uint8_t *sbuf_dev) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= kIterNum) return;

  uint8_t *sbuf = sbuf_dev + tid * kLambda * 6;
  uint8_t *cw_np1 = cw_np1_dev + tid * kLambda;
  uint8_t *cws = cws_dev + tid * kDpfCwLen * kAlphaBitlen;

  DpfKey k = {cws, cw_np1};
  PointFunc pf = {{alpha_dev, kAlphaBitlen}, beta_dev};
  dpf_gen(k, pf, sbuf);
}

__global__ void eval_kernel(
  uint8_t *sbuf_dev, uint8_t b, const uint8_t *cw_np1_dev, const uint8_t *cws_dev, const uint8_t *x_dev) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= kIterNum) return;

  uint8_t *sbuf = sbuf_dev + tid * kLambda * 3;
  const uint8_t *x = x_dev + tid * sizeof(uint16_t);
  const uint8_t *cw_np1 = cw_np1_dev + tid * kLambda;
  const uint8_t *cws = cws_dev + tid * kDpfCwLen * kAlphaBitlen;

  DpfKey k = {(uint8_t *)cws, (uint8_t *)cw_np1};
  Bits x_bits = {x, kAlphaBitlen};
  dpf_eval(sbuf, b, k, x_bits);
}

void cudaAssert(hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    assert(false);
  }
}

int main() {
  srand(kSeed);
  assert(kLambda == 16);
  double t;
  hipError_t err;

  uint32_t nonce[2] = {0, 1};
  prg_init((const uint8_t *)nonce, 8);

  uint16_t alpha_int = kAlpha;
  uint8_t *alpha_dev;
  err = hipMalloc(&alpha_dev, sizeof(uint16_t));
  cudaAssert(err);
  err = hipMemcpy(alpha_dev, &alpha_int, sizeof(uint16_t), hipMemcpyHostToDevice);
  cudaAssert(err);
  __uint128_t beta_int = kBeta;
  uint8_t *beta_dev;
  err = hipMalloc(&beta_dev, sizeof(__uint128_t));
  cudaAssert(err);
  err = hipMemcpy(beta_dev, &beta_int, sizeof(__uint128_t), hipMemcpyHostToDevice);
  cudaAssert(err);

  uint8_t *sbuf_dev;
  err = hipMalloc(&sbuf_dev, kLambda * 6 * kIterNum);
  cudaAssert(err);
  uint8_t *cw_np1_dev;
  err = hipMalloc(&cw_np1_dev, kLambda * kIterNum);
  cudaAssert(err);
  uint8_t *cws_dev;
  err = hipMalloc(&cws_dev, kDpfCwLen * kAlphaBitlen * kIterNum);
  cudaAssert(err);

  uint8_t *s0s = (uint8_t *)malloc(kLambda * 2);
  assert(s0s != NULL);
  memset(s0s, 0, kLambda * 2);
  s0s[0] = 114;
  s0s[kLambda - 1] = 51;
  s0s[kLambda] = 4;
  for (int i = 0; i < kIterNum; i++) {
    err = hipMemcpy(sbuf_dev + i * kLambda * 6, s0s, kLambda * 2, hipMemcpyHostToDevice);
    cudaAssert(err);
  }

  t = get_time();
  int block_num = (kIterNum + kThreadsPerBlock - 1) / kThreadsPerBlock;
  gen_kernel<<<block_num, kThreadsPerBlock>>>(cw_np1_dev, cws_dev, alpha_dev, beta_dev, sbuf_dev);
  hipDeviceSynchronize();
  printf("dpf_gen (s/%d): %lf\n", kIterNum, get_time() - t);

  err = hipFree(alpha_dev);
  cudaAssert(err);
  err = hipFree(beta_dev);
  cudaAssert(err);
  err = hipFree(sbuf_dev);
  cudaAssert(err);

  uint16_t *x_int = (uint16_t *)malloc(sizeof(uint16_t) * kIterNum);
  assert(x_int != NULL);
  for (int i = 0; i < kIterNum; i++) {
    if (i == 0) {
      x_int[i] = kAlpha;
    } else {
      x_int[i] = rand() & UINT16_MAX;
    }
  }

  err = hipMalloc(&sbuf_dev, kLambda * 3 * kIterNum);
  cudaAssert(err);

  uint8_t *x_dev;
  err = hipMalloc(&x_dev, sizeof(uint16_t) * kIterNum);
  cudaAssert(err);
  err = hipMemcpy(x_dev, x_int, sizeof(uint16_t) * kIterNum, hipMemcpyHostToDevice);
  cudaAssert(err);

  __uint128_t *y0_int = (__uint128_t *)malloc(kLambda * kIterNum);
  assert(y0_int != NULL);
  __uint128_t *y1_int = (__uint128_t *)malloc(kLambda * kIterNum);
  assert(y1_int != NULL);

  uint8_t *sbuf_tmp = (uint8_t *)malloc(kLambda * 3 * kIterNum);
  assert(sbuf_tmp != NULL);
  for (int i = 0; i < kIterNum; i++) {
    memcpy(sbuf_tmp + i * kLambda * 3, s0s, kLambda);
  }

  t = get_time();
  err = hipMemcpy(sbuf_dev, sbuf_tmp, kLambda * 3 * kIterNum, hipMemcpyHostToDevice);
  cudaAssert(err);
  eval_kernel<<<block_num, kThreadsPerBlock>>>(sbuf_dev, 0, cw_np1_dev, cws_dev, x_dev);
  hipDeviceSynchronize();
  err = hipMemcpy(sbuf_tmp, sbuf_dev, kLambda * 3 * kIterNum, hipMemcpyDeviceToHost);
  cudaAssert(err);
  printf("dpf_eval (s/%d): %lf\n", kIterNum, get_time() - t);

  for (int i = 0; i < kIterNum; i++) {
    y0_int[i] = *(__uint128_t *)(sbuf_tmp + i * kLambda * 3);
  }

  for (int i = 0; i < kIterNum; i++) {
    memcpy(sbuf_tmp + i * kLambda * 3, s0s + kLambda, kLambda);
  }

  err = hipMemcpy(sbuf_dev, sbuf_tmp, kLambda * 3 * kIterNum, hipMemcpyHostToDevice);
  cudaAssert(err);
  eval_kernel<<<block_num, kThreadsPerBlock>>>(sbuf_dev, 1, cw_np1_dev, cws_dev, x_dev);
  hipDeviceSynchronize();
  err = hipMemcpy(sbuf_tmp, sbuf_dev, kLambda * 3 * kIterNum, hipMemcpyDeviceToHost);
  cudaAssert(err);
  for (int i = 0; i < kIterNum; i++) {
    y1_int[i] = *(__uint128_t *)(sbuf_tmp + i * kLambda * 3);
  }

  for (int i = 0; i < kIterNum; i++) {
    __uint128_t y_int = *(y0_int + i) + *(y1_int + i);
    if (x_int[i] == kAlpha) {
      assert(y_int == kBeta);
    } else {
      assert(y_int == 0);
    }
  }

  err = hipFree(cw_np1_dev);
  cudaAssert(err);
  err = hipFree(cws_dev);
  cudaAssert(err);
  err = hipFree(sbuf_dev);
  cudaAssert(err);
  err = hipFree(x_dev);
  cudaAssert(err);
  free(s0s);
  free(x_int);
  free(y0_int);
  free(y1_int);
  free(sbuf_tmp);
  return 0;
}
